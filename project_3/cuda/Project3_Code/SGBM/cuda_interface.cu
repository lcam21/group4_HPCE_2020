// CUDA-C includes
#include <hip/hip_runtime.h>

#include <stdio.h>

__device__ int find_minLri(int *Lrpr, uint16_t m_u16TotalDisp) {
    int minLri = INT_MAX;
    for (int i=0; i<m_u16TotalDisp; i++) {
        if (minLri > Lrpr[i]) {
            minLri = Lrpr[i];
        }
    }
    return minLri;
} // end find_minLri()

__device__ int find_min(int a, int b, int c, int d) {
    int minimum = a;
    if (minimum > b)
        minimum = b;
    if (minimum > c)
        minimum = c;
    if (minimum > d)
        minimum = d;
    return minimum;
} // end find_min()

__global__ void cost_computation(int *Lr, int *initCost, uint8_t m_u8Directions, uint8_t m_u8DEBUGLeftNeighbor,
                                 uint16_t m_u16height_after_census, uint16_t m_u16width_after_census,
                                 uint16_t m_u16TotalDisp, uint8_t m_u16P1, uint8_t m_u16P2) {
    //#######################################################################################
    //          SGBM Cost Computation
    //#######################################################################################

    int r = blockIdx.x * blockDim.x + threadIdx.x;

    // Computing cost along 5 directions only. (i,j-1) (i-1,j-1) (i-1,j) (i-1,j+1) (i,j+1)
    __shared__ int iDisp, jDisp;

    if (r < m_u8Directions) {
    //for (int r=0; r<m_u8Directions; r++) {
        if (r==0) {
            iDisp = 0; jDisp = 0-m_u8DEBUGLeftNeighbor;
        }
        else if (r==1) {
            iDisp = -1; jDisp = -1;
        }
        else if (r==2) {
            iDisp = -1; jDisp = 0;
        }
        else if (r==3) {
            iDisp = -1; jDisp = 1;
        }
        else if (r==4) {
            iDisp = 0; jDisp = 1;
        }
        //if (i < m_u16height_after_census) {
        for (int i=0; i<m_u16height_after_census; i++) {
            //if (i < m_u16width_after_census) {
            for (int j=0; j<m_u16width_after_census; j++) {
                // Compute p-r
                int iNorm = i + iDisp; //height
                int jNorm = j + jDisp; //width
                int *Lrpr = Lr+((r*m_u16height_after_census+iNorm)*m_u16width_after_census+jNorm)*m_u16TotalDisp;

                //#######################################################################################
                //         Find min_k{Lr(p-r,k)}
                //#######################################################################################
                //if (d < m_u16TotalDisp) {
                for (int d=0; d<m_u16TotalDisp; d++) {
                    int Cpd = initCost[(i*m_u16width_after_census+j)*m_u16TotalDisp+d];

                    int tmp;
                    if ( (((r==0)||(r==1))&&(jNorm<0)) || (((r==1)||(r==2)||(r==3))&&(i==0)) || ((r==3)&&(j==m_u16width_after_census-1)))
                    {
                        tmp = Cpd;
                    } else {
                        int minLri = find_minLri(Lrpr, m_u16TotalDisp);
                        int Lrpdm1, Lrpdp1;
                        if (d==0)
                            Lrpdm1 = INT_MAX-m_u16P1;
                        else
                            Lrpdm1 = Lrpr[d-1];
                        if (d==m_u16TotalDisp-1)
                            Lrpdp1 = INT_MAX-m_u16P1;
                        else
                            Lrpdp1 = Lrpr[d+1];

                        int v1 = find_min(Lrpr[d], Lrpdm1+m_u16P1, Lrpdp1+m_u16P1, minLri+m_u16P2);

                        tmp = Cpd + v1 - minLri;
                    }
                    Lr[((r*m_u16height_after_census+i)*m_u16width_after_census+j)*m_u16TotalDisp+d] = tmp;
                }
            }
        }
    }
} // end cost_computation()

__device__ int compute_hamming_distance (uint64_t a, uint64_t b)
{
    //#######################################################################################
    //Hamming Distance as cost initialization
    //#######################################################################################
    uint64_t tmp;
    int sum = 0;

    tmp = a^b;
    for(int i=0;i<64;i++){
        if(tmp&0x1<<i){
            sum ++;
        }
    }
    return sum;
} // end compute_hamming_distance()



__global__ void compute_hamming(uint64_t *ct1, uint64_t *ct2, int *accumulatedCost,
                                uint64_t m_u16yMin, uint64_t m_u16yMax, uint64_t m_u16xMin, uint64_t m_u16xMax,
                                uint64_t m_u16TotalDisp, uint64_t m_u16width_after_census, uint64_t m_s16MinDisp)
{
    //#######################################################################################
    //Hamming Distance as cost initialization
    //#######################################################################################

    int d = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i=m_u16yMin; i<m_u16yMax; i++) {
        for (int j=m_u16xMin; j<m_u16xMax; j++) {
            //for (int d=0; d<m_u16TotalDisp; d++) {
            if (d < m_u16TotalDisp) {
                int dist = compute_hamming_distance(ct1[i*m_u16width_after_census+j], ct2[i*m_u16width_after_census+j-(d+m_s16MinDisp)]);
                int Loop = (i*m_u16width_after_census+j)*m_u16TotalDisp + d;
                accumulatedCost[Loop] = dist;
            }
        }
    }
} // end compute_cost()


__global__ void cost_aggregation(int *aggregatedCost, int *Lr, uint16_t m_u16height_after_census,
                                 uint16_t m_u16width_after_census, uint16_t m_u16TotalDisp, uint16_t m_u8Directions)
{
    //#######################################################################################
    //          SGBM Cost Aggregation
    //#######################################################################################
    for (int i=0; i<m_u16height_after_census; i++)
    {
        for (int j=0; j<m_u16width_after_census; j++)
        {
            for (int d=0; d<m_u16TotalDisp; d++)
            {
                int *ptr = aggregatedCost + (i*m_u16width_after_census+j)*m_u16TotalDisp+d;
                ptr[0] = 0;

                for (int r=0; r<m_u8Directions; r++)
                {
                    ptr[0] += Lr[((r*m_u16height_after_census+i)*m_u16width_after_census+j)*m_u16TotalDisp+d];
                }
            }
        }
    }
}// end cost_aggregation()
